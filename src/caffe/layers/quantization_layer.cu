#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include <math.h>

#include "caffe/layers/quantization_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void QTForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = round(in[index]);
  }
}

template <typename Dtype>
void QuantizationLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  QTForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void QuantizationLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    caffe_copy(bottom[0]->count(),top_diff,bottom_diff);
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(QuantizationLayer);


}  // namespace caffe
